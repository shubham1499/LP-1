
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <string>
#define TOGG(k) ( ( ( (k) & 1 ) << 5 ) )
#define RAN(charac) ( 65 + ( charac % 26 ) + TOGG ( charac ) ) 
using namespace std;

__global__ 
void RunLengthEncodingComputation (char *orig, int *_encoXst, int n) {
    int index = ( (blockIdx.x * blockDim.x) + threadIdx.x );
    index <<= 7;

    if(orig[index] == orig[index-1]) 
        while(index < n && orig[index] == orig[index-1])
            ++index;
    
    for (int i = index; i < fminf(index + 128, n); )
    {
        char temp = orig[i];
        int t_ = i;
        while (i < n && temp == orig[i])
            ++i;
        _encoXst[t_] = i;
    }
}

int main()  {
    int n;
    cin >> n;
    char *s = new char[n];
    int i = 0;

    for( ; i < n; ) {
        char in = RAN ( rand() );
        int loop = rand() & 63, k = i;
        while(i < min(k+loop, n))
            s[i++] = in;
    }
    s[i] = '\0';
    fprintf(fopen("input.txt", "w"), "%s", s);
    char *cudas;
    int *_encoXst, *_inter = new int[n];

    int threads = (1 << 7);
    int blocks = ( ( n>>14 ) + ( ( n & ( (1<<14)-1 ) ) != 0 )  );

    cout << threads << " : " << blocks << endl;

    hipMalloc (&cudas, n*sizeof(char));
    hipMalloc (&cudas, n*sizeof(char));
    hipMalloc (&_encoXst, n*sizeof(int));

    hipMemcpy (cudas, s, n*sizeof(char), hipMemcpyHostToDevice);

    RunLengthEncodingComputation <<<blocks, threads>>> (cudas, _encoXst, n);
    hipDeviceSynchronize();
    hipMemcpy(_inter, _encoXst, n*sizeof(int), hipMemcpyDeviceToHost);

    string ans;
    int sum = 0;
    for(int i = 0; i < n; i = _inter[i]) {
        ans += s[i] + to_string(_inter[i]-i);
        sum += _inter[i]-i;
    }
    // cout << ans << endl;
    fprintf(fopen("output.txt", "w"), "%s", ans.c_str());
    int length_ans = ans.length();
    printf("Length: %d\nCompressed Length: %d\nCompression Achievement: %f\n", sum, length_ans, (float)(sum+length_ans-1)/(length_ans));
}